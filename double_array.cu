#include "hip/hip_runtime.h"
#include <stdio.h>

// Utility to pretty-print arrays
void print_array(int *arr, int size) {
    for(int i=0; i<size; i++)
        printf("%d ", arr[i]);
    printf("\n");
}

// Kernel that doubles each element in an array and stores the result
__global__ void double_arr(int *input, int *output) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    output[index] = input[index] * 2;
}

int main(int argc, char *argv[]) {
    int problem_size = 10;
    int input[problem_size] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
    int output[problem_size];

    // 1. Allocate GPU Memory
    int *d_output, *d_input;
    hipMalloc(&d_output, sizeof(int)*problem_size);
    hipMalloc(&d_input, sizeof(int)*problem_size);

    // 2. Copy input array to d_input
    hipMemcpy(d_input, input, sizeof(int)*problem_size, hipMemcpyHostToDevice);

    // 3. Run kernel using 1 block of 10 threads
    double_arr<<<1, 10>>>(d_input, d_output);

    // 4. Copy d_output back to output
    hipMemcpy(output, d_output, sizeof(int)*problem_size, hipMemcpyDeviceToHost);

    // 5. Print out the array and check the result
    print_array(output, problem_size);
}
