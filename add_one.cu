#include "hip/hip_runtime.h"
#include <stdio.h>

// Pretty-print arr of size size
void print_array(int *arr, int size) {
    for(int i=0; i<size; i++)
        printf("%d ", arr[i]);
    printf("\n");
}

// The GPU kernel - each thread adds one to a given element
__global__ void addOne(int *input, int *output) {
    // Calculate global index based on thread position variables
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    output[index] = input[index] + 1;
}

int main(int argc, char *argv[]) {
    int problem_size = 8;
    int input[problem_size] = {0, 1, 2, 3, 4, 5, 6, 7};
    int output[problem_size];

    // GPU Memory Allocation
    int *d_input, *d_output;
    hipMalloc(&d_input, sizeof(int)*problem_size);
    hipMalloc(&d_output, sizeof(int)*problem_size);

    // Copy data from input (CPU array) to d_input (GPU array)
    hipMemcpy(d_input, input, sizeof(int)*problem_size, hipMemcpyHostToDevice);

    // Launch kernel with 2 blocks of 4 threads
    addOne<<<2, 4>>>(d_input, d_output);
  
    // Copy data from d_output (GPU array) to output (CPU array)
    hipMemcpy(output, d_output, sizeof(int)*problem_size, hipMemcpyDeviceToHost);

    // Print out result to check for correct result
    print_array(output, problem_size);
}
